#include "hip/hip_runtime.h"
#include "../include/fastRayTriangleIntersection_parallel.cuh"

__device__ void cross(double* a, double* b, double* result) {
	result[0] = a[1] * b[2] - a[2] * b[1];
	result[1] = a[2] * b[0] - a[0] * b[2];
	result[2] = a[0] * b[1] - a[1] * b[0];
}

__global__ void kernel_fastRayTriangleIntersection(
	double orig[COLUMNS_SIZE], double dir[COLUMNS_SIZE],
	double** V1, double** V2, double** V3, unsigned short rows,
	unsigned short border, unsigned short lineType, unsigned short planeType, bool fullReturn,
	bool* intersect, double* t, double* u, double* v) 
{
	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (row < rows) {
		double eps = 1e-5, zero;
		
		switch (border) {
			case BORDER_NORMAL:
				zero = 0.0;
				break;
			case BORDER_INCLUSIVE:
				zero = eps;
				break;
			case BORDER_EXCLUSIVE:
				zero = -eps;
				break;
			//default:
				//TODO: Handle error
		}
		
		__shared__ double edge1[ROWS_SIZE][COLUMNS_SIZE], edge2[ROWS_SIZE][COLUMNS_SIZE],
			tvec[ROWS_SIZE][COLUMNS_SIZE], pvec[ROWS_SIZE][COLUMNS_SIZE], det[ROWS_SIZE];
		
		for (unsigned short i = 0; i < COLUMNS_SIZE; i++) {
			edge1[row][i] = V2[row][i] - V1[row][i];
			edge2[row][i] = V3[row][i] - V1[row][i];
			tvec[row][i] = orig[i] - V1[row][i];
		}
		
		pvec[row][0] = dir[1] * edge2[row][2] - dir[2] * edge2[row][1];
		pvec[row][1] = dir[2] * edge2[row][0] - dir[0] * edge2[row][2];
		pvec[row][2] = dir[0] * edge2[row][1] - dir[1] * edge2[row][0];
		
		det[row] = 0;
		for (unsigned short i = 0; i < COLUMNS_SIZE; i++)
			det[row] += edge1[row][i] * pvec[row][i];
		
		if (planeType == PLANE_TYPE_TWOSIDED)
			intersect[row] = abs(det[row]) > eps;
		else if (planeType == PLANE_TYPE_ONESIDED)
			intersect[row] = det[row] > eps;
		else {
			// Handle error
			return;
		}
		
		if (!intersect[row])
			u[row] = NAN;
		else {
			u[row] = 0;
			for (unsigned short i = 0; i < COLUMNS_SIZE; i++)
				u[row] += tvec[row][i] * pvec[row][i];
			
			u[row] /= det[row];
		}
		
		//__syncthreads();

		if (fullReturn) {
			__shared__ double qvec[COLUMNS_SIZE];
			if (!intersect[row])
				v[row] = NAN, t[row] = NAN;
			else {
				qvec[0] = tvec[row][1] * edge1[row][2] - tvec[row][2] * edge1[row][1];
				qvec[1] = tvec[row][2] * edge1[row][0] - tvec[row][0] * edge1[row][2];
				qvec[2] = tvec[row][0] * edge1[row][1] - tvec[row][1] * edge1[row][0];
				
				v[row] = t[row] = 0;
				for (unsigned short i = 0; i < COLUMNS_SIZE; i++){
					v[row] += dir[i] * qvec[i];
					t[row] += edge2[row][i] * qvec[i];
				}
				
				v[row] /= det[row];
				t[row] /= det[row];
				
				intersect[row] = u[row] >= -zero && v[row] >= -zero && u[row] + v[row] <= 1.0 + zero;
			}

			//__syncthreads();

		} else {
			__shared__ double qvec[COLUMNS_SIZE];

			intersect[row] = intersect[row] && u[row] >= -zero && u[row] <= 1 + zero;

			if (!intersect[row])
				v[row] = NAN;
			else {
				qvec[0] = tvec[row][1] * edge1[row][2] - tvec[row][2] * edge1[row][1];
				qvec[1] = tvec[row][2] * edge1[row][0] - tvec[row][0] * edge1[row][2];
				qvec[2] = tvec[row][0] * edge1[row][1] - tvec[row][1] * edge1[row][0];
				
				v[row] = 0;
				for (unsigned short i = 0; i < COLUMNS_SIZE; i++)
					v[row] += dir[i] * qvec[i];
				v[row] /= det[row];
				
				if (lineType == LINE_TYPE_LINE)
					t[row] = NAN;
				else {
					t[row] = 0;
					for (unsigned short i = 0; i < COLUMNS_SIZE; i++)
						t[row] += edge2[row][i] * qvec[i];
					t[row] /= det[row];
				}
				
				intersect[row] = v[row] >= -zero && u[row] + v[row] <= 1.0 + zero;
			}

			//__syncthreads();
		}
	
		switch (lineType) {
		case LINE_TYPE_LINE:// Nothing to do
			break;
		case LINE_TYPE_RAY:
				intersect[row] = intersect[row] && t[row] >= -zero;
			break;
		case LINE_TYPE_SEGMENT:
				intersect[row] = intersect[row] && t[row] >= -zero && t[row] <= 1.0 + zero;
			break;
		//default:
			//TODO: std::cerr << "LineType parameter must be either 'line', 'ray' or 'segment'\n";
		}
	}
	
}