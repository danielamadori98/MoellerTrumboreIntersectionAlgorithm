#include "hip/hip_runtime.h"
#include "../include/check_visibility_parallel.cuh"

bool* check_visibility_parallel_code(
	double* camera_location,
	double** verts, unsigned short verts_rows,
	unsigned short** meshes, unsigned short meshes_rows,
	unsigned short columns,
	double** V1, double** V2, double** V3,
	bool* flag,
	double* t, double* u, double* v, bool* visible)
{
	// Creating the device variables
	double* d_camera_location, * d_verts,
		** d_meshes, ** d_V1, ** d_V2, ** d_V3,
		* d_t, * d_u, * d_v;
	bool* d_flag;

	// Initialize CUDA
	hipError_t cudaStatus = hipSetDevice(0);  // You can set the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int device;
	hipGetDevice(&device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	unsigned short maxStreams = deviceProp.asyncEngineCount;
	std::cout << "Maximum number of CUDA streams on GPU " << device << ": " << maxStreams << std::endl;

	// Allocate memory for device variables and copy data from host to device
	hipMalloc((void**)&d_camera_location, columns * sizeof(double));
	hipMalloc((void**)&d_verts, verts_rows * columns * sizeof(double));
	hipMalloc((void**)&d_meshes, meshes_rows * columns * sizeof(unsigned short));
	hipMalloc((void**)&d_V1, meshes_rows * columns * sizeof(double));
	hipMalloc((void**)&d_V2, meshes_rows * columns * sizeof(double));
	hipMalloc((void**)&d_V3, meshes_rows * columns * sizeof(double));
	
	hipMalloc((void**)&d_flag, meshes_rows * sizeof(bool));
	hipMalloc((void**)&d_t, meshes_rows * sizeof(double));
	hipMalloc((void**)&d_u, meshes_rows * sizeof(double));
	hipMalloc((void**)&d_v, meshes_rows * sizeof(double));

	std::cout << "After Allocation\n";

	// Creating the streams
	unsigned short segSize = 32, segNumber = (unsigned short)std::round(meshes_rows / segSize);
	dim3 blockDim(segSize, 1);
	dim3 gridDim(segNumber, 1);


	hipStream_t* meshes_streams = new hipStream_t[maxStreams];
	for (unsigned short i = 0; i < maxStreams; i++)
		hipStreamCreate(&meshes_streams[i]);

	for (unsigned short i = 0; i < verts_rows; i++) {
		visible[i] = true;

		for (unsigned short j = 0; j < segNumber && visible[i]; j++) {
			unsigned short stream = j % maxStreams;

			hipMemcpyAsync(d_camera_location, camera_location, columns * sizeof(double), hipMemcpyHostToDevice, meshes_streams[stream]);
			hipMemcpyAsync(d_verts + i, verts[i], columns * sizeof(double), hipMemcpyHostToDevice, meshes_streams[stream]);			
			hipMemcpyAsync(d_meshes + j * segSize, meshes + j * segSize, segSize * columns * sizeof(unsigned short), hipMemcpyHostToDevice, meshes_streams[stream]);
			hipMemcpyAsync(d_V1 + j * segSize, V1 + j * segSize, segSize * columns * sizeof(double), hipMemcpyHostToDevice, meshes_streams[stream]);
			hipMemcpyAsync(d_V2 + j * segSize, V2 + j * segSize, segSize * columns * sizeof(double), hipMemcpyHostToDevice, meshes_streams[stream]);
			hipMemcpyAsync(d_V3 + j * segSize, V3 + j * segSize, segSize * columns * sizeof(double), hipMemcpyHostToDevice, meshes_streams[stream]);
			
			//std::cout << "After copying all, j = " << j << "\n";

			kernel_fastRayTriangleIntersection << <gridDim, blockDim, 0, meshes_streams[stream] >> >
				(d_camera_location, d_verts + i,
					d_V1 + j * segSize, d_V2 + j * segSize, d_V3 + j * segSize,
					segSize,
					BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED, false,
					d_flag + j, d_t + j, d_u + j, d_v + j
					);

			//std::cout << "After kernel, j = " << j << "\n";

			hipMemcpyAsync(flag + j * segSize, d_flag + j * segSize, segSize * sizeof(bool), hipMemcpyDeviceToHost, meshes_streams[stream]);
			hipMemcpyAsync(t + j * segSize, d_t + j * segSize, segSize * sizeof(double), hipMemcpyDeviceToHost, meshes_streams[stream]);
			hipMemcpyAsync(u + j * segSize, d_u + j * segSize, segSize * sizeof(double), hipMemcpyDeviceToHost, meshes_streams[stream]);
			hipMemcpyAsync(v + j * segSize, d_v + j * segSize, segSize * sizeof(double), hipMemcpyDeviceToHost, meshes_streams[stream]);
			
			for (unsigned short k = j * segSize; k < j * segSize + segSize; k++)
				if (flag[k]) {
					visible[i] = false;
					break;
				}

			//std::cout << "After checking visibility, visible[" << i << "] = " << visible[i] << "\n";
		}
	}

	//Free memory
	hipFree(d_camera_location);
	hipFree(d_verts);
	hipFree(d_meshes);
	hipFree(d_V1);
	hipFree(d_V2);
	hipFree(d_V3);

	hipFree(d_flag);
	hipFree(d_t);
	hipFree(d_u);
	hipFree(d_v);

	std::cout << "After freeing all\n";

	return visible;
}
