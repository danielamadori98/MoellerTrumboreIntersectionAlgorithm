#include "hip/hip_runtime.h"
#include "../include/check_visibility_parallel.cuh"


void copy_verts_to_device(double** verts, unsigned short verts_rows, double* d_vert) {
	double h_vert[COLUMNS_SIZE];

	for (unsigned short verts_row = 0, d_V_row = 0; verts_row < verts_rows; verts_row++, d_V_row += COLUMNS_SIZE) {
		for (unsigned short col = 0; col < COLUMNS_SIZE; col++)
			h_vert[col] = verts[verts_row][col];

		hipMemcpy(d_vert + d_V_row, h_vert, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	}
}

//Zero error
double check_visibility_parallel_code(
	double camera_location[COLUMNS_SIZE],
	double** verts, unsigned short verts_rows,
	double* h_V1, double* h_V2, double* h_V3, unsigned short V_rows,
	bool* flag, double* t, double* u, double* v, bool* visible) // Output variables
{
	hipError_t cudaStatus = hipSetDevice(1);  // Setting the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int device;
	hipGetDevice(&device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	unsigned short streams_size = deviceProp.asyncEngineCount;

	hipStream_t* streams = new hipStream_t[streams_size];
	for (unsigned short i = 0; i < streams_size; i++)
		hipStreamCreate(&streams[i]);

	// Creating the device variables
	double* d_camera_location, * d_vert,
		* d_V1, * d_V2, * d_V3,
		* d_t, * d_u, * d_v;

	bool* d_flag;
	unsigned short* d_visible, * h_visible; // Using h_visible to copy back from device to host (Pinned memory)
	hipHostMalloc((void**)&h_visible, verts_rows * sizeof(unsigned short));

	hipMalloc((void**)&d_visible, verts_rows * sizeof(unsigned short));

	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_vert, verts_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_flag, BLOCK_ROWS_SIZE * sizeof(bool));
	hipMalloc((void**)&d_t, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_u, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_v, BLOCK_ROWS_SIZE * sizeof(double));


	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	
	hipMemcpy(d_V1, h_V1, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V2, h_V2, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V3, h_V3, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	
	copy_verts_to_device(verts, verts_rows, d_vert);

	for(unsigned short verts_row = 0; verts_row < verts_rows; verts_row++)
		h_visible[verts_row] = 0;
	hipMemcpy(d_visible, h_visible, verts_rows * sizeof(unsigned short), hipMemcpyHostToDevice);


	timer::Timer<timer::DEVICE> dev_TM;
	dev_TM.start();

	for (unsigned short verts_row = 0, d_verts_row = 0; verts_row < verts_rows; verts_row++, d_verts_row += COLUMNS_SIZE) {

		unsigned short V_row, d_V_row;
		for (V_row = 0, d_V_row = 0; V_row < V_rows; V_row += BLOCK_ROWS_SIZE, d_V_row += V_BLOCK_SIZE) {
			//std::cout << "V_row = " << V_row << "\n";

			fastRayTriangleIntersection_parallel_with_check << <gridDim, blockDim, 0>> > (
				d_camera_location, d_vert + d_verts_row,
				d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, BLOCK_ROWS_SIZE,
				BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED, false,
				d_flag, d_t, d_u, d_v,
				d_visible + verts_row);	
		}

		fastRayTriangleIntersection_parallel_with_check << <gridDim, blockDim, 0 >> > (
			d_camera_location, d_vert + d_verts_row,
			d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, V_rows - V_row,
			BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED, false,
			d_flag, d_t, d_u, d_v,
			d_visible + verts_row);
	}

	dev_TM.stop();
	dev_TM.print("MoellerTrumboreIntersectionAlgorithm device:   ");


	hipMemcpy(h_visible, d_visible, verts_rows * sizeof(unsigned short), hipMemcpyDeviceToHost);

	for(unsigned short i = 0; i < verts_rows; i++)
		visible[i] = h_visible[i] == 0;

	hipFree(d_camera_location), hipFree(d_vert);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_flag), hipFree(d_t), hipFree(d_u), hipFree(d_v);
	hipFree(d_visible);
	hipHostFree(h_visible);

	return dev_TM.duration();
}


//Zero error (sometimes)
void check_visibility_parallel_code_streams(
	double camera_location[COLUMNS_SIZE],
	double** verts, unsigned short verts_rows,
	double* h_V1, double* h_V2, double* h_V3, unsigned short V_rows,
	bool* flag, double* t, double* u, double* v, bool* visible) // Output variables
{
	hipError_t cudaStatus = hipSetDevice(1);  // Setting the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int device;
	hipGetDevice(&device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	unsigned short streams_size = deviceProp.asyncEngineCount;

	hipStream_t* streams = new hipStream_t[streams_size];
	for (unsigned short i = 0; i < streams_size; i++)
		hipStreamCreate(&streams[i]);

	// Creating the device variables
	double* d_camera_location, * d_vert,
		* d_V1, * d_V2, * d_V3,
		* d_t, * d_u, * d_v;

	bool* d_flag;
	unsigned short *d_visible, *h_visible; // Using h_visible to copy back from device to host (Pinned memory)
	hipHostMalloc((void**)&h_visible, streams_size * sizeof(unsigned short));

	hipMalloc((void**)&d_visible, sizeof(unsigned short));

	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_vert, verts_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_flag, BLOCK_ROWS_SIZE * sizeof(bool));
	hipMalloc((void**)&d_t, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_u, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_v, BLOCK_ROWS_SIZE * sizeof(double));


	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(d_V1, h_V1, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V2, h_V2, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V3, h_V3, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);

	copy_verts_to_device(verts, verts_rows, d_vert);

	for (unsigned short verts_row = 0, d_verts_row = 0; verts_row < verts_rows; verts_row++, d_verts_row += COLUMNS_SIZE) {
		visible[verts_row] = true;
		
		for (unsigned short V_row = 0, d_V_row = 0; V_row < V_rows;) {
			//std::cout << "V_row = " << V_row << "\n";

			for (unsigned short stream = 0; stream < streams_size; stream++, V_row += BLOCK_ROWS_SIZE, d_V_row += V_BLOCK_SIZE) {
				unsigned short remaing_rows = V_rows - V_row;
				if (remaing_rows >= BLOCK_ROWS_SIZE)
					remaing_rows = BLOCK_ROWS_SIZE;
				
				fastRayTriangleIntersection_parallel_with_check << <gridDim, blockDim, 0, streams[stream] >> > (
					d_camera_location, d_vert + d_verts_row,
					d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, remaing_rows,
					BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED, false,
					d_flag, d_t, d_u, d_v,
					d_visible);

				/*
				hipMemcpyAsync(flag + V_row, d_flag, BLOCK_ROWS_SIZE * sizeof(bool), hipMemcpyDeviceToHost, streams[stream]);
				hipMemcpyAsync(t + V_row, d_t, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);
				hipMemcpyAsync(u + V_row, d_u, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);
				hipMemcpyAsync(v + V_row, d_v, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);
				*/
				hipMemcpyAsync(h_visible + stream, d_visible, sizeof(unsigned short), hipMemcpyDeviceToHost, streams[stream]);
				//std::cout << "h_visible = " << h_visible[stream] << "\n";
				//hipStreamSynchronize(streams[stream]);
			}
			// TODO: fix cudaStreamSyncronize work just if you put it twice: in the strems loop and after it (IDK why)
			for(unsigned short stream = 0; stream < streams_size; stream++){
				hipStreamSynchronize(streams[stream]);
				if (h_visible[stream] > 0) 
					visible[verts_row] = false;
			}
		}

		if (verts_row % 100 == 0)
			std::cout << "Visible[" << verts_row << "] = " << visible[verts_row] << "\n";
	}

	hipFree(d_camera_location), hipFree(d_vert);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_flag), hipFree(d_t), hipFree(d_u), hipFree(d_v);
	hipFree(d_visible);
	hipHostFree(h_visible);
}


