#include "hip/hip_runtime.h"
#include "../include/check_visibility_parallel.cuh"


double check_visibility_parallel_code(
	double camera_location[COLUMNS_SIZE],
	double* h_verts, unsigned short verts_rows,
	double* h_V1, double* h_V2, double* h_V3, unsigned short V_rows,
	bool* flag, double* t, double* u, double* v, bool* visible) // Output variables
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0) {
		std::cerr << "No CUDA devices found." << std::endl;
		return -1;
	}

	int device = 1; // You can change this to the desired GPU device index
	hipError_t cudaStatus = hipSetDevice(device);  // Setting the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int sharedMemoryPerBlock;
	hipDeviceGetAttribute(&sharedMemoryPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device);

	//std::cout << "Shared Memory per Block on GPU " << device << ": " << sharedMemoryPerBlock << " bytes" << std::endl;

	std::cout << "Max Teorical limit of BLOCK_ROWS_SIZE given by shared memory is: "
		<< MAX_BLOCK_ROWS_SIZE(sharedMemoryPerBlock / MAX_SPACE_COST_FULL_RETURN, 64)
		<< "\nYou are using a BLOCK_ROWS_SIZE of: " << BLOCK_ROWS_SIZE << std::endl;

	double* d_camera_location, * d_verts,
		* d_V1, * d_V2, * d_V3,
		* d_t, * d_u, * d_v;

	bool* d_flag;

	unsigned int* d_visible, * h_visible; // Using h_visible to copy back from device to host (Pinned memory)

	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_verts, verts_rows * COLUMNS_SIZE * sizeof(double));
	hipHostMalloc((void**)&h_visible, verts_rows * sizeof(unsigned int));
	hipMalloc((void**)&d_visible, verts_rows * sizeof(unsigned int));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_flag, BLOCK_ROWS_SIZE * sizeof(bool));
	hipMalloc((void**)&d_t, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_u, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_v, BLOCK_ROWS_SIZE * sizeof(double));

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	
	hipMemcpy(d_verts, h_verts, verts_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	for (unsigned short verts_row = 0; verts_row < verts_rows; verts_row++)
		h_visible[verts_row] = 0;
	hipMemcpy(d_visible, h_visible, verts_rows * sizeof(unsigned int), hipMemcpyHostToDevice);

	hipMemcpy(d_V1, h_V1, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V2, h_V2, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V3, h_V3, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	

	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);


	timer::Timer<timer::DEVICE> dev_TM;
	dev_TM.start();

	unsigned short V_row, d_V_row;
	for (unsigned short verts_row = 0, d_verts_row = 0; verts_row < verts_rows; verts_row++, d_verts_row += COLUMNS_SIZE) {
		for (V_row = 0, d_V_row = 0; V_row < V_rows; V_row += BLOCK_ROWS_SIZE, d_V_row += BLOCK_ROWS_SIZE * COLUMNS_SIZE) {
			fastRayTriangleIntersection_parallel<< <gridDim, blockDim, 0>> > (
				d_camera_location, d_verts + d_verts_row,
				d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, BLOCK_ROWS_SIZE,
				BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED,
				d_flag, d_t, d_u, d_v,
				d_visible + verts_row);

			hipMemcpy(flag + V_row, d_flag, BLOCK_ROWS_SIZE * sizeof(bool), hipMemcpyDeviceToHost);
			hipMemcpy(t + V_row, d_t, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(u + V_row, d_u, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(v + V_row, d_v, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
		}

		fastRayTriangleIntersection_parallel<< <gridDim, blockDim, 0 >> > (
			d_camera_location, d_verts + d_verts_row,
			d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, V_rows - V_row,
			BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED,
			d_flag, d_t, d_u, d_v,
			d_visible + verts_row);

		hipMemcpy(flag + V_row, d_flag, BLOCK_ROWS_SIZE * sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(t + V_row, d_t, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(u + V_row, d_u, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(v + V_row, d_v, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
	}

	dev_TM.stop();
	dev_TM.print("MoellerTrumboreIntersectionAlgorithm device:   ");


	hipMemcpy(h_visible, d_visible, verts_rows * sizeof(unsigned int), hipMemcpyDeviceToHost);

	for(unsigned short i = 0; i < verts_rows; i++)
		visible[i] = h_visible[i] == 0;

	hipFree(d_camera_location), hipFree(d_verts);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_flag), hipFree(d_t), hipFree(d_u), hipFree(d_v);
	hipFree(d_visible);
	hipHostFree(h_visible);

	return dev_TM.duration();
}


double check_visibility_parallel_code(
	double camera_location[COLUMNS_SIZE],
	double* h_verts, unsigned short verts_rows,
	double* h_V1, double* h_V2, double* h_V3, unsigned short V_rows,
	bool* visible) // Output variable
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0) {
		std::cerr << "No CUDA devices found." << std::endl;
		return -1;
	}

	int device = 1; // You can change this to the desired GPU device index
	hipError_t cudaStatus = hipSetDevice(device);  // Setting the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int sharedMemoryPerBlock;
	hipDeviceGetAttribute(&sharedMemoryPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device);

	//std::cout << "Shared Memory per Block on GPU " << device << ": " << sharedMemoryPerBlock << " bytes" << std::endl;

	std::cout << "Max Teorical limit of BLOCK_ROWS_SIZE given by shared memory is: "
		<< MAX_BLOCK_ROWS_SIZE(sharedMemoryPerBlock / MAX_SPACE_COST, 64)
		<< "\nYou are using a BLOCK_ROWS_SIZE of: " << BLOCK_ROWS_SIZE << std::endl;

	double* d_camera_location, * d_verts,
		* d_V1, * d_V2, * d_V3;

	unsigned int* d_visible, * h_visible; // Using h_visible to copy back from device to host (Pinned memory)

	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_verts, verts_rows * COLUMNS_SIZE * sizeof(double));
	hipHostMalloc((void**)&h_visible, verts_rows * sizeof(unsigned int));
	hipMalloc((void**)&d_visible, verts_rows * sizeof(unsigned int));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(d_verts, h_verts, verts_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	for (unsigned short verts_row = 0; verts_row < verts_rows; verts_row++)
		h_visible[verts_row] = 0;
	hipMemcpy(d_visible, h_visible, verts_rows * sizeof(unsigned int), hipMemcpyHostToDevice);

	hipMemcpy(d_V1, h_V1, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V2, h_V2, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V3, h_V3, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);


	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);


	timer::Timer<timer::DEVICE> dev_TM;
	dev_TM.start();

	unsigned short V_row, d_V_row;
	for (unsigned short verts_row = 0, d_verts_row = 0; verts_row < verts_rows; verts_row++, d_verts_row += COLUMNS_SIZE) {
		for (V_row = 0, d_V_row = 0; V_row < V_rows; V_row += BLOCK_ROWS_SIZE, d_V_row += BLOCK_ROWS_SIZE * COLUMNS_SIZE) {
			fastRayTriangleIntersection_parallel << <gridDim, blockDim, 0 >> > (
				d_camera_location, d_verts + d_verts_row,
				d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, BLOCK_ROWS_SIZE,
				BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED,
				d_visible + verts_row);
		}

		fastRayTriangleIntersection_parallel << <gridDim, blockDim, 0 >> > (
			d_camera_location, d_verts + d_verts_row,
			d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, V_rows - V_row,
			BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED,
			d_visible + verts_row);
	}

	dev_TM.stop();
	dev_TM.print("MoellerTrumboreIntersectionAlgorithm device:   ");


	hipMemcpy(h_visible, d_visible, verts_rows * sizeof(unsigned int), hipMemcpyDeviceToHost);

	for (unsigned short i = 0; i < verts_rows; i++)
		visible[i] = h_visible[i] == 0;

	hipFree(d_camera_location), hipFree(d_verts);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_visible);
	hipHostFree(h_visible);

	return dev_TM.duration();
}
