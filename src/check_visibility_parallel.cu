#include "hip/hip_runtime.h"
#include "../include/check_visibility_parallel.cuh"

#include "../include/lib/CheckError.cuh"


double check_visibility_parallel_code(
	double camera_location[COLUMNS_SIZE],
	double* h_verts, unsigned short verts_rows,
	double* h_V1, double* h_V2, double* h_V3, unsigned short V_rows,
	bool* flag, double* t, double* u, double* v, bool* visible) // Output variables
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0) {
		std::cerr << "No CUDA devices found." << std::endl;
		return -1;
	}

	int device = 1; // You can change this to the desired GPU device index
	hipError_t cudaStatus = hipSetDevice(device);  // Setting the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int sharedMemoryPerBlock;
	hipDeviceGetAttribute(&sharedMemoryPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device);

	//std::cout << "Shared Memory per Block on GPU " << device << ": " << sharedMemoryPerBlock << " bytes" << std::endl;

	std::cout << "Max Teorical limit of BLOCK_ROWS_SIZE given by shared memory is: "
		<< MAX_BLOCK_ROWS_SIZE(sharedMemoryPerBlock / MAX_SPACE_COST_FULL_RETURN, 64)
		<< "\nYou are using a BLOCK_ROWS_SIZE of: " << BLOCK_ROWS_SIZE << std::endl;

	double* d_camera_location, * d_verts,
		* d_V1, * d_V2, * d_V3,
		* d_t, * d_u, * d_v;

	bool* d_flag;

	unsigned int* d_visible, * h_visible; // Using h_visible to copy back from device to host (Pinned memory)

	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_verts, verts_rows * COLUMNS_SIZE * sizeof(double));
	hipHostMalloc((void**)&h_visible, verts_rows * sizeof(unsigned int));
	hipMalloc((void**)&d_visible, verts_rows * sizeof(unsigned int));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_flag, BLOCK_ROWS_SIZE * sizeof(bool));
	hipMalloc((void**)&d_t, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_u, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_v, BLOCK_ROWS_SIZE * sizeof(double));

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	
	hipMemcpy(d_verts, h_verts, verts_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	for (unsigned short verts_row = 0; verts_row < verts_rows; verts_row++)
		h_visible[verts_row] = 0;
	hipMemcpy(d_visible, h_visible, verts_rows * sizeof(unsigned int), hipMemcpyHostToDevice);

	hipMemcpy(d_V1, h_V1, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V2, h_V2, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V3, h_V3, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	

	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);


	timer::Timer<timer::DEVICE> dev_TM;
	dev_TM.start();

	unsigned short V_row, d_V_row;
	for (unsigned short verts_row = 0, d_verts_row = 0; verts_row < verts_rows; verts_row++, d_verts_row += COLUMNS_SIZE) {
		for (V_row = 0, d_V_row = 0; V_row < V_rows; V_row += BLOCK_ROWS_SIZE, d_V_row += BLOCK_ROWS_SIZE * COLUMNS_SIZE) {
			fastRayTriangleIntersection_parallel<< <gridDim, blockDim, 0>> > (
				d_camera_location, d_verts + d_verts_row,
				d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, BLOCK_ROWS_SIZE,
				BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED,
				d_flag, d_t, d_u, d_v,
				d_visible + verts_row);

			hipMemcpy(flag + V_row, d_flag, BLOCK_ROWS_SIZE * sizeof(bool), hipMemcpyDeviceToHost);
			hipMemcpy(t + V_row, d_t, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(u + V_row, d_u, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(v + V_row, d_v, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
		}

		fastRayTriangleIntersection_parallel<< <gridDim, blockDim, 0 >> > (
			d_camera_location, d_verts + d_verts_row,
			d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, V_rows - V_row,
			BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED,
			d_flag, d_t, d_u, d_v,
			d_visible + verts_row);

		hipMemcpy(flag + V_row, d_flag, BLOCK_ROWS_SIZE * sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(t + V_row, d_t, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(u + V_row, d_u, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(v + V_row, d_v, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
	}

	dev_TM.stop();
	dev_TM.print("MoellerTrumboreIntersectionAlgorithm device:   ");


	hipMemcpy(h_visible, d_visible, verts_rows * sizeof(unsigned int), hipMemcpyDeviceToHost);

	for(unsigned short i = 0; i < verts_rows; i++)
		visible[i] = h_visible[i] == 0;

	hipFree(d_camera_location), hipFree(d_verts);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_flag), hipFree(d_t), hipFree(d_u), hipFree(d_v);
	hipFree(d_visible);
	hipHostFree(h_visible);

	return dev_TM.duration();
}


double check_visibility_parallel_code(
	double camera_location[COLUMNS_SIZE],
	double* h_verts, unsigned short verts_rows,
	double* h_V1, double* h_V2, double* h_V3, unsigned short V_rows,
	bool* visible) // Output variable
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0) {
		std::cerr << "No CUDA devices found." << std::endl;
		return -1;
	}

	int device = 1; // You can change this to the desired GPU device index
	hipError_t cudaStatus = hipSetDevice(device);  // Setting the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int sharedMemoryPerBlock;
	hipDeviceGetAttribute(&sharedMemoryPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device);

	//std::cout << "Shared Memory per Block on GPU " << device << ": " << sharedMemoryPerBlock << " bytes" << std::endl;

	std::cout << "Max Teorical limit of BLOCK_ROWS_SIZE given by shared memory is: "
		<< MAX_BLOCK_ROWS_SIZE(sharedMemoryPerBlock / MAX_SPACE_COST, 64)
		<< "\nYou are using a BLOCK_ROWS_SIZE of: " << BLOCK_ROWS_SIZE << std::endl;

	double* d_camera_location, * d_verts,
		* d_V1, * d_V2, * d_V3;

	unsigned int* d_visible, * h_visible; // Using h_visible to copy back from device to host (Pinned memory)

	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_verts, verts_rows * COLUMNS_SIZE * sizeof(double));
	hipHostMalloc((void**)&h_visible, verts_rows * sizeof(unsigned int));
	hipMalloc((void**)&d_visible, verts_rows * sizeof(unsigned int));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(d_verts, h_verts, verts_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	for (unsigned short verts_row = 0; verts_row < verts_rows; verts_row++)
		h_visible[verts_row] = 0;
	hipMemcpy(d_visible, h_visible, verts_rows * sizeof(unsigned int), hipMemcpyHostToDevice);

	hipMemcpy(d_V1, h_V1, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V2, h_V2, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V3, h_V3, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);


	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);


	timer::Timer<timer::DEVICE> dev_TM;
	dev_TM.start();

	unsigned short V_row, d_V_row;
	for (unsigned short verts_row = 0, d_verts_row = 0; verts_row < verts_rows; verts_row++, d_verts_row += COLUMNS_SIZE) {
		for (V_row = 0, d_V_row = 0; V_row < V_rows; V_row += BLOCK_ROWS_SIZE, d_V_row += BLOCK_ROWS_SIZE * COLUMNS_SIZE) {
			fastRayTriangleIntersection_parallel << <gridDim, blockDim, 0 >> > (
				d_camera_location, d_verts + d_verts_row,
				d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, BLOCK_ROWS_SIZE,
				BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED,
				d_visible + verts_row);
		}

		fastRayTriangleIntersection_parallel << <gridDim, blockDim, 0 >> > (
			d_camera_location, d_verts + d_verts_row,
			d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, V_rows - V_row,
			BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED,
			d_visible + verts_row);
	}

	dev_TM.stop();
	dev_TM.print("MoellerTrumboreIntersectionAlgorithm device:   ");


	hipMemcpy(h_visible, d_visible, verts_rows * sizeof(unsigned int), hipMemcpyDeviceToHost);

	for (unsigned short i = 0; i < verts_rows; i++)
		visible[i] = h_visible[i] == 0;

	hipFree(d_camera_location), hipFree(d_verts);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_visible);
	hipHostFree(h_visible);

	return dev_TM.duration();
}



double check_visibility_parallel_code_dyn(
	double camera_location[COLUMNS_SIZE],
	double* h_verts, unsigned short verts_rows,
	double* h_V1, double* h_V2, double* h_V3, unsigned short V_rows,
	bool* visible) // Output variable
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0) {
		std::cerr << "No CUDA devices found." << std::endl;
		return -1;
	}

	int device = 1; // You can change this to the desired GPU device index
	hipError_t cudaStatus = hipSetDevice(device);  // Setting the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int sharedMemoryPerBlock, maxThreadsPerBlock;
	hipDeviceGetAttribute(&sharedMemoryPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
	hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, device);

	//std::cout << "Shared Memory per Block on GPU " << device << ": " << sharedMemoryPerBlock << " bytes" << std::endl;
	const unsigned short max_block_rows_size = MAX_BLOCK_ROWS_SIZE(sharedMemoryPerBlock / MAX_SPACE_COST, 64);
	const unsigned short shared_memory_cost = max_block_rows_size * MAX_SPACE_COST;
	std::cout << "Max limit of threads per block is: " << maxThreadsPerBlock << " you are using " << BLOCK_ROWS_SIZE
		<< "\nMax limit of BLOCK_ROWS_SIZE given by shared memory is: " << max_block_rows_size
		<< "\nYou are using " << shared_memory_cost << " / " << sharedMemoryPerBlock << " Bytes of shared mem\n";

	double* d_camera_location, * d_verts,
		* d_V1, * d_V2, * d_V3;

	unsigned int* d_visible, * h_visible; // Using h_visible to copy back from device to host (Pinned memory)

	SAFE_CALL(hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double)));

	SAFE_CALL(hipMalloc((void**)&d_verts, verts_rows * COLUMNS_SIZE * sizeof(double)));
	SAFE_CALL(hipHostMalloc((void**)&h_visible, verts_rows * sizeof(unsigned int)));
	SAFE_CALL(hipMalloc((void**)&d_visible, verts_rows * sizeof(unsigned int)));

	SAFE_CALL(hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double)));
	SAFE_CALL(hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double)));
	SAFE_CALL(hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double)));

	SAFE_CALL(hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice));

	SAFE_CALL(hipMemcpy(d_verts, h_verts, verts_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice));
	for (unsigned short verts_row = 0; verts_row < verts_rows; verts_row++)
		h_visible[verts_row] = 0;
	SAFE_CALL(hipMemcpy(d_visible, h_visible, verts_rows * sizeof(unsigned int), hipMemcpyHostToDevice));

	SAFE_CALL(hipMemcpy(d_V1, h_V1, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(d_V2, h_V2, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(d_V3, h_V3, V_rows * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice));


	dim3 blockDim(1, max_block_rows_size);
	dim3 gridDim(std::ceil(V_rows / max_block_rows_size), 1);


	timer::Timer<timer::DEVICE> dev_TM;
	dev_TM.start();

	unsigned short V_row, d_V_row;
	for (unsigned short verts_row = 0, d_verts_row = 0; verts_row < 1/*verts_rows*/; verts_row++, d_verts_row += COLUMNS_SIZE) {
		for (V_row = 0, d_V_row = 0; V_row < 1/*V_rows*/; V_row += max_block_rows_size, d_V_row += max_block_rows_size * COLUMNS_SIZE) {
			fastRayTriangleIntersection_parallel_dyn << <gridDim, blockDim, shared_memory_cost >> > (
				d_camera_location, d_verts + d_verts_row,
				d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, max_block_rows_size,
				max_block_rows_size,
				BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED,
				d_visible + verts_row);

			CHECK_CUDA_ERROR
		}
		
		fastRayTriangleIntersection_parallel_dyn << <gridDim, blockDim, shared_memory_cost >> > (
			d_camera_location, d_verts + d_verts_row,
			d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, V_row- V_rows,
			max_block_rows_size,
			BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED,
			d_visible + verts_row);
		
		CHECK_CUDA_ERROR
		
	}
	

	dev_TM.stop();
	dev_TM.print("MoellerTrumboreIntersectionAlgorithm device:   ");


	SAFE_CALL(hipMemcpy(h_visible, d_visible, verts_rows * sizeof(unsigned int), hipMemcpyDeviceToHost));

	for (unsigned short i = 0; i < verts_rows; i++)
		visible[i] = h_visible[i] == 0;

	SAFE_CALL(hipFree(d_camera_location));
	SAFE_CALL(hipFree(d_verts));
	SAFE_CALL(hipFree(d_V1));
	SAFE_CALL(hipFree(d_V2));
	SAFE_CALL(hipFree(d_V3));
	SAFE_CALL(hipFree(d_visible));
	SAFE_CALL(hipHostFree(h_visible));

	return dev_TM.duration();
}
