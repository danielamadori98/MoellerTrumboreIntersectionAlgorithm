#include "hip/hip_runtime.h"
#include "../include/check_visibility_parallel.cuh"


void copy_V_arrays_to_device(double** V1, double** V2, double** V3, unsigned short V_rows,
	double* d_V1, double* d_V2, double* d_V3) {

	double h_V1[V_BLOCK_SIZE], h_V2[V_BLOCK_SIZE], h_V3[V_BLOCK_SIZE];

	unsigned short V_row, d_V_row;
	for (V_row = 0, d_V_row = 0; V_row < V_rows - BLOCK_ROWS_SIZE + 1; V_row += BLOCK_ROWS_SIZE, d_V_row += V_BLOCK_SIZE) {
		for (unsigned short row = 0; row < BLOCK_ROWS_SIZE; row++)
			for (unsigned short col = 0; col < COLUMNS_SIZE; col++) {
				h_V1[row * COLUMNS_SIZE + col] = V1[V_row + row][col];
				h_V2[row * COLUMNS_SIZE + col] = V2[V_row + row][col];
				h_V3[row * COLUMNS_SIZE + col] = V3[V_row + row][col];
			}

		hipMemcpy(d_V1 + d_V_row, h_V1, V_BLOCK_SIZE * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_V2 + d_V_row, h_V2, V_BLOCK_SIZE * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_V3 + d_V_row, h_V3, V_BLOCK_SIZE * sizeof(double), hipMemcpyHostToDevice);
	}
	unsigned short V_last_row_size = V_rows - V_row;
	for (unsigned short row = 0; row < V_last_row_size; row++)
		for (unsigned short col = 0; col < COLUMNS_SIZE; col++) {
			h_V1[row * COLUMNS_SIZE + col] = V1[V_row + row][col];
			h_V2[row * COLUMNS_SIZE + col] = V2[V_row + row][col];
			h_V3[row * COLUMNS_SIZE + col] = V3[V_row + row][col];
		}
	hipMemcpy(d_V1 + d_V_row, h_V1, V_last_row_size * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V2 + d_V_row, h_V2, V_last_row_size * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V3 + d_V_row, h_V3, V_last_row_size * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
}

void copy_verts_to_device(double** verts, unsigned short verts_rows, double* d_vert) {
	double h_vert[COLUMNS_SIZE];

	for (unsigned short verts_row = 0, d_V_row = 0; verts_row < verts_rows; verts_row++, d_V_row += COLUMNS_SIZE) {
		for (unsigned short col = 0; col < COLUMNS_SIZE; col++)
			h_vert[col] = verts[verts_row][col];

		hipMemcpy(d_vert + d_V_row, h_vert, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	}
}


//Zero error
void check_visibility_parallel_code_with_check(
	double camera_location[COLUMNS_SIZE],
	double** verts, unsigned short verts_rows,
	double** V1, double** V2, double** V3, unsigned short V_rows,
	bool* flag, double* t, double* u, double* v, bool* visible) // Output variables
{
	hipError_t cudaStatus = hipSetDevice(1);  // Setting the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int device;
	hipGetDevice(&device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	unsigned short streams_size = deviceProp.asyncEngineCount;

	hipStream_t* streams = new hipStream_t[streams_size];
	for (unsigned short i = 0; i < streams_size; i++)
		hipStreamCreate(&streams[i]);

	// Creating the device variables
	double* d_camera_location, * d_vert,
		* d_V1, * d_V2, * d_V3,
		* d_t, * d_u, * d_v;

	bool* d_flag;
	unsigned short *d_visible, *h_visible; // Using h_visible to copy back from device to host (Pinned memory)
	hipHostMalloc((void**)&h_visible, streams_size * sizeof(unsigned short));

	hipMalloc((void**)&d_visible, sizeof(unsigned short));

	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_vert, verts_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_flag, BLOCK_ROWS_SIZE * sizeof(bool));
	hipMalloc((void**)&d_t, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_u, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_v, BLOCK_ROWS_SIZE * sizeof(double));


	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	copy_V_arrays_to_device(V1, V2, V3, V_rows, d_V1, d_V2, d_V3);
	copy_verts_to_device(verts, verts_rows, d_vert);

	for (unsigned short verts_row = 0, d_verts_row = 0; verts_row < verts_rows; verts_row++, d_verts_row += COLUMNS_SIZE) {
		visible[verts_row] = true;
		
		for (unsigned short V_row = 0, d_V_row = 0; V_row < V_rows;) {
			//std::cout << "V_row = " << V_row << "\n";

			for (unsigned short stream = 0; stream < streams_size; stream++, V_row += BLOCK_ROWS_SIZE, d_V_row += V_BLOCK_SIZE) {
				unsigned short remaing_rows = V_rows - V_row;
				if (remaing_rows >= BLOCK_ROWS_SIZE)
					remaing_rows = BLOCK_ROWS_SIZE;
				
				fastRayTriangleIntersection_parallel_with_check << <gridDim, blockDim, 0, streams[stream] >> > (
					d_camera_location, d_vert + d_verts_row,
					d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, remaing_rows,
					BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED, false,
					d_flag, d_t, d_u, d_v,
					d_visible);

				/*
				hipMemcpyAsync(flag + V_row, d_flag, BLOCK_ROWS_SIZE * sizeof(bool), hipMemcpyDeviceToHost, streams[stream]);
				hipMemcpyAsync(t + V_row, d_t, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);
				hipMemcpyAsync(u + V_row, d_u, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);
				hipMemcpyAsync(v + V_row, d_v, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);
				*/
				hipMemcpyAsync(h_visible + stream, d_visible, sizeof(unsigned short), hipMemcpyDeviceToHost, streams[stream]);
				//std::cout << "h_visible = " << h_visible[stream] << "\n";
				hipStreamSynchronize(streams[stream]);
			}
			// TODO: fix cudaStreamSyncronize work inside streams loop but not after it
			
			for(unsigned short stream = 0; stream < streams_size; stream++){
				hipStreamSynchronize(streams[stream]);
				if (h_visible[stream] > 0) 
					visible[verts_row] = false;
			}
		}

		if (verts_row % 100 == 0)
			std::cout << "Visible[" << verts_row << "] = " << visible[verts_row] << "\n";
	}

	hipFree(d_camera_location), hipFree(d_vert);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_flag), hipFree(d_t), hipFree(d_u), hipFree(d_v);
	hipFree(d_visible);
	hipHostFree(h_visible);
}


void check_visibility_parallel_code(
	double camera_location[COLUMNS_SIZE],
	double** verts, unsigned short verts_rows,
	double** V1, double** V2, double** V3, unsigned short V_rows,
	bool* flag, double* t, double* u, double* v, bool* visible) // Output variables
{
	// Initialize CUDA
	hipError_t cudaStatus = hipSetDevice(1);  // You can set the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	// Creating the device variables
	double* d_camera_location, * d_vert,
		* d_V1, * d_V2, * d_V3,
		* d_t, * d_u, * d_v;

	bool* d_flag;

	// Allocate memory for device variables and copy data from host to device
	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_vert, verts_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_flag, BLOCK_ROWS_SIZE * sizeof(bool));
	hipMalloc((void**)&d_t, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_u, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_v, BLOCK_ROWS_SIZE * sizeof(double));
	

	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	copy_V_arrays_to_device(V1, V2, V3, V_rows, d_V1, d_V2, d_V3);
	copy_verts_to_device(verts, verts_rows, d_vert);

	for (unsigned short verts_row = 0, d_verts_row = 0; verts_row < verts_rows; verts_row++, d_verts_row += COLUMNS_SIZE) {
		visible[verts_row] = true;

		for (unsigned short V_row = 0, d_V_row = 0; V_row < V_rows - BLOCK_ROWS_SIZE + 1 && visible[verts_row]; V_row += BLOCK_ROWS_SIZE, d_V_row += V_BLOCK_SIZE) {
			//std::cout << "V_row = " << V_row << "\n";

			fastRayTriangleIntersection_parallel << <gridDim, blockDim, 0 >> > (
				d_camera_location, d_vert + d_verts_row,
				d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, BLOCK_ROWS_SIZE,
				BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED, false,
				d_flag, d_t, d_u, d_v);

			//std::cout << "After kernel, V_row = " << V_row << "\n";
			hipMemcpy(flag + V_row, d_flag, BLOCK_ROWS_SIZE * sizeof(bool), hipMemcpyDeviceToHost);
			/*
			hipMemcpy(t + V_row, d_t, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(u + V_row, d_u, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(v + V_row, d_v, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
			*/

			hipDeviceSynchronize(); //TODO check if needed (it seems that it is needed)
			for(unsigned short row = 0; row < BLOCK_ROWS_SIZE; row++)
				if (flag[V_row + row]) {
					visible[row] = false;
					break;
				}
		}
		if (verts_row % 100 == 0)
			std::cout << "Visible[" << verts_row << "] = " << visible[verts_row] << "\n";
	}

	hipFree(d_camera_location), hipFree(d_vert);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_flag), hipFree(d_t), hipFree(d_u), hipFree(d_v);
}


void check_visibility_parallel_code_streams(
	double camera_location[COLUMNS_SIZE],
	double** verts, unsigned short verts_rows,
	double** V1, double** V2, double** V3, unsigned short V_rows,
	bool* flag, double* t, double* u, double* v, bool* visible) // Output variables
{
	// Initialize CUDA
	hipError_t cudaStatus = hipSetDevice(1);  // You can set the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int device;
	hipGetDevice(&device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	unsigned short streams_size = deviceProp.asyncEngineCount;

	hipStream_t* streams = new hipStream_t[streams_size];
	for (unsigned short i = 0; i < streams_size; i++)
		hipStreamCreate(&streams[i]);

	// Creating the device variables
	double* d_camera_location, * d_vert,
		* d_V1, * d_V2, * d_V3,
		* d_t, * d_u, * d_v;

	bool* d_flag;

	// Allocate memory for device variables and copy data from host to device
	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_vert, verts_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_flag, BLOCK_ROWS_SIZE * sizeof(bool));
	hipMalloc((void**)&d_t, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_u, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_v, BLOCK_ROWS_SIZE * sizeof(double));

	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	copy_V_arrays_to_device(V1, V2, V3, V_rows, d_V1, d_V2, d_V3);
	copy_verts_to_device(verts, verts_rows, d_vert);

	unsigned short stream = 0;
	for (unsigned short verts_row = 0, d_verts_row = 0; verts_row < verts_rows; verts_row++, d_verts_row += COLUMNS_SIZE) {
		visible[verts_row] = true;
		stream = verts_row % streams_size;

		for (unsigned short V_row = 0, d_V_row = 0; V_row < V_rows - BLOCK_ROWS_SIZE + 1; V_row += BLOCK_ROWS_SIZE, d_V_row += V_BLOCK_SIZE) {
			//std::cout << "V_row = " << V_row << "\n";

			fastRayTriangleIntersection_parallel<<<gridDim, blockDim, 0, streams[stream]>>>(
				d_camera_location, d_vert + d_verts_row,
				d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, BLOCK_ROWS_SIZE,
				BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED, false,
				d_flag, d_t, d_u, d_v);

			//std::cout << "After kernel, V_row = " << V_row << "\n";

			hipMemcpyAsync(flag + V_row, d_flag, BLOCK_ROWS_SIZE * sizeof(bool), hipMemcpyDeviceToHost, streams[stream]);
			//hipMemcpyAsync(t + V_row, d_t, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);
			//hipMemcpyAsync(u + V_row, d_u, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);
			//hipMemcpyAsync(v + V_row, d_v, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);

			//std::cout << "After copying back all, V_row = " << V_row << "\n";

			for (unsigned short k = V_row; k < V_row + BLOCK_ROWS_SIZE; k++)
				if (flag[k]) {
					visible[verts_row] = false;
					break;
				}
		}

		if (verts_row % 100 == 0)
			std::cout << "Visible[" << verts_row << "] = " << visible[verts_row] << "\n";
	}

	hipFree(d_camera_location), hipFree(d_vert);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_flag), hipFree(d_t), hipFree(d_u), hipFree(d_v);
}
