#include "hip/hip_runtime.h"
#include "../include/check_visibility_parallel.cuh"

bool* check_visibility_parallel_code(
	double* camera_location,
	double** verts, unsigned short verts_rows,
	unsigned short** meshes, unsigned short meshes_rows,
	unsigned short columns,
	double** V1, double** V2, double** V3,
	bool* flag,
	double* t, double* u, double* v, bool* visible)
{
	// Creating the device variables
	double* d_camera_location, * d_verts,
		** d_meshes, ** d_V1, ** d_V2, ** d_V3,
		* d_t, * d_u, * d_v;
	bool* d_flag, * d_visible;

	// Initialize CUDA
	hipError_t cudaStatus = hipSetDevice(0);  // You can set the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;


	int device;
	hipGetDevice(&device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	unsigned short maxStreams = deviceProp.asyncEngineCount;
	std::cout << "Maximum number of CUDA streams on GPU " << device << ": " << maxStreams << std::endl;


	std::cout << "Before Allocation\n";

	// Allocate memory for device variables and copy data from host to device
	hipMalloc((void**)&d_camera_location, columns * sizeof(double));
	hipMalloc((void**)&d_verts, verts_rows * columns * sizeof(double));

	hipMalloc((void**)&d_meshes, meshes_rows * columns * sizeof(unsigned short));
	hipMalloc((void**)&d_V1, meshes_rows * columns * sizeof(double));
	hipMalloc((void**)&d_V2, meshes_rows * columns * sizeof(double));
	hipMalloc((void**)&d_V3, meshes_rows * columns * sizeof(double));
	hipMalloc((void**)&d_flag, meshes_rows * sizeof(bool));

	hipMalloc((void**)&d_t, meshes_rows * sizeof(double));
	hipMalloc((void**)&d_u, meshes_rows * sizeof(double));
	hipMalloc((void**)&d_v, meshes_rows * sizeof(double));
	hipMalloc((void**)&d_visible, meshes_rows * sizeof(bool));

	std::cout << "After Allocation\n";

	// Creating the streams
	unsigned short segSize = 32, segNumber = (unsigned short)std::round(meshes_rows / segSize);
	dim3 blockDim(segSize, 1);
	dim3 gridDim(segNumber, 1);


	hipStream_t* meshes_streams = new hipStream_t[maxStreams];
	for (unsigned short i = 0; i < maxStreams; i++)
		hipStreamCreate(&meshes_streams[i]);

	for (unsigned short i = 0; i < verts_rows; i++) {
		visible[i] = true;

		for (unsigned short j = 0; j < segNumber && visible[i]; j++) {
			unsigned short stream = j % maxStreams;
			hipMemcpyAsync(d_camera_location, camera_location, columns * sizeof(double), hipMemcpyHostToDevice, meshes_streams[stream]);
			//std::cout << "After copyng camera_location\n";
			hipMemcpyAsync(d_verts + i, verts[i], columns * sizeof(double), hipMemcpyHostToDevice, meshes_streams[stream]);
			//std::cout << "After copying verts[" << i << "]\n";
			
			hipMemcpyAsync(d_meshes + j * segSize, meshes + j * segSize, segSize * columns * sizeof(unsigned short), hipMemcpyHostToDevice, meshes_streams[stream]);
			hipMemcpyAsync(d_V1 + j * segSize, V1 + j * segSize, segSize * columns * sizeof(double), hipMemcpyHostToDevice, meshes_streams[stream]);
			hipMemcpyAsync(d_V2 + j * segSize, V2 + j * segSize, segSize * columns * sizeof(double), hipMemcpyHostToDevice, meshes_streams[stream]);
			hipMemcpyAsync(d_V3 + j * segSize, V3 + j * segSize, segSize * columns * sizeof(double), hipMemcpyHostToDevice, meshes_streams[stream]);
			//std::cout << "After copying all, j = " << j << "\n";

			kernel_fastRayTriangleIntersection << <gridDim, blockDim, 0, meshes_streams[stream] >> >
				(d_camera_location, d_verts + i,
					d_V1 + j * segSize, d_V2 + j * segSize, d_V3 + j * segSize,
					segSize,
					BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED, false,
					d_flag + j, d_t + j, d_u + j, d_v + j
					);

			//std::cout << "After kernel, j = " << j << "\n";

			hipMemcpyAsync(flag + j * segSize, d_flag + j * segSize, segSize * sizeof(bool), hipMemcpyDeviceToHost, meshes_streams[stream]);
			//std::cout << "After flag, j = " << j << "\n";
			hipMemcpyAsync(t + j * segSize, d_t + j * segSize, segSize * sizeof(double), hipMemcpyDeviceToHost, meshes_streams[stream]);
			//std::cout << "After t, j = " << j << "\n";
			hipMemcpyAsync(u + j * segSize, d_u + j * segSize, segSize * sizeof(double), hipMemcpyDeviceToHost, meshes_streams[stream]);
			//std::cout << "After u, j = " << j << "\n";
			hipMemcpyAsync(v + j * segSize, d_v + j * segSize, segSize * sizeof(double), hipMemcpyDeviceToHost, meshes_streams[stream]);
			//std::cout << "After v, j = " << j << "\n";
			
			for (unsigned short k = j * segSize; k < j * segSize + segSize; k++)
				if (flag[k]) {
					visible[i] = false;
					break;
				}
		}
	}

	std::cout << "After kernel\n";

	//Free memory
	hipFree(d_camera_location);

	std::cout << "After freeing d_camera_location\n";

	/*delete[] d_camera_location;
	delete[] d_verts;
	delete[] d_V1;
	delete[] d_V2;
	delete[] d_V3;
	delete[] d_flag;
	delete[] d_t;
	delete[] d_u;
	delete[] d_v;
	delete[] d_visible;
	*/
	std::cout << "After freeing all\n";

	return visible;
}
