#include "hip/hip_runtime.h"
#include "../include/check_visibility_parallel.cuh"


void copy_V_arrays_to_device(double** V1, double** V2, double** V3, unsigned short V_rows,
	double* d_V1, double* d_V2, double* d_V3) {

	double h_V1[V_BLOCK_SIZE], h_V2[V_BLOCK_SIZE], h_V3[V_BLOCK_SIZE];

	unsigned short V_row, d_V_row;
	for (V_row = 0, d_V_row = 0; V_row < V_rows - BLOCK_ROWS_SIZE + 1; V_row += BLOCK_ROWS_SIZE, d_V_row += V_BLOCK_SIZE) {
		for (unsigned short row = 0; row < BLOCK_ROWS_SIZE; row++)
			for (unsigned short col = 0; col < COLUMNS_SIZE; col++) {
				h_V1[row * COLUMNS_SIZE + col] = V1[V_row + row][col];
				h_V2[row * COLUMNS_SIZE + col] = V2[V_row + row][col];
				h_V3[row * COLUMNS_SIZE + col] = V3[V_row + row][col];
			}

		hipMemcpy(d_V1 + d_V_row, h_V1, V_BLOCK_SIZE * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_V2 + d_V_row, h_V2, V_BLOCK_SIZE * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_V3 + d_V_row, h_V3, V_BLOCK_SIZE * sizeof(double), hipMemcpyHostToDevice);
	}
	unsigned short V_last_row_size = V_rows - V_row;
	for (unsigned short row = 0; row < V_last_row_size; row++)
		for (unsigned short col = 0; col < COLUMNS_SIZE; col++) {
			h_V1[row * COLUMNS_SIZE + col] = V1[V_row + row][col];
			h_V2[row * COLUMNS_SIZE + col] = V2[V_row + row][col];
			h_V3[row * COLUMNS_SIZE + col] = V3[V_row + row][col];
		}
	hipMemcpy(d_V1 + d_V_row, h_V1, V_last_row_size * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V2 + d_V_row, h_V2, V_last_row_size * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V3 + d_V_row, h_V3, V_last_row_size * COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
}



void check_visibility_parallel_code(
	double camera_location[COLUMNS_SIZE],
	double** verts, unsigned short verts_rows,
	double** V1, double** V2, double** V3, unsigned short V_rows,
	bool* flag, double* t, double* u, double* v, bool* visible) // Output variables
{
	// Initialize CUDA
	hipError_t cudaStatus = hipSetDevice(1);  // You can set the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	// Creating the device variables
	double* d_camera_location, * d_vert,
		* d_V1, * d_V2, * d_V3,
		* d_t, * d_u, * d_v;

	bool* d_flag;

	// Allocate memory for device variables and copy data from host to device
	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_vert, COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_flag, BLOCK_ROWS_SIZE * sizeof(bool));
	hipMalloc((void**)&d_t, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_u, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_v, BLOCK_ROWS_SIZE * sizeof(double));

	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	copy_V_arrays_to_device(V1, V2, V3, V_rows, d_V1, d_V2, d_V3);

	for (unsigned short verts_row = 0; verts_row < verts_rows; verts_row++) {
		visible[verts_row] = true;

		hipMemcpy(d_vert, verts[verts_row], COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
		for (unsigned short V_row = 0, d_V_row = 0; V_row < V_rows - BLOCK_ROWS_SIZE + 1; V_row += BLOCK_ROWS_SIZE, d_V_row += V_BLOCK_SIZE) {
			//std::cout << "V_row = " << V_row << "\n";

			fastRayTriangleIntersection_parallel << <gridDim, blockDim, 0 >> > (
				d_camera_location, d_vert,
				d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, BLOCK_ROWS_SIZE,
				BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED, false,
				d_flag, d_t, d_u, d_v);

			hipDeviceSynchronize();

			//std::cout << "After kernel, V_row = " << V_row << "\n";

			hipMemcpy(flag + V_row, d_flag, BLOCK_ROWS_SIZE * sizeof(bool), hipMemcpyDeviceToHost);
			//hipMemcpy(t + V_row, d_t, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
			//hipMemcpy(u + V_row, d_u, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);
			//hipMemcpy(v + V_row, d_v, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost);

			//std::cout << "After copying back all, V_row = " << V_row << "\n";

			for (unsigned short k = V_row; k < V_row + BLOCK_ROWS_SIZE; k++)
				if (flag[k]) {
					visible[verts_row] = false;
					break;
				}
		}
		if (verts_row % 100 == 0)
			std::cout << "Visible[" << verts_row << "] = " << visible[verts_row] << "\n";
	}

	hipFree(d_camera_location), hipFree(d_vert);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_flag), hipFree(d_t), hipFree(d_u), hipFree(d_v);
}


void check_visibility_parallel_code_streams(
	double camera_location[COLUMNS_SIZE],
	double** verts, unsigned short verts_rows,
	double** V1, double** V2, double** V3, unsigned short V_rows,
	bool* flag, double* t, double* u, double* v, bool* visible) // Output variables
{
	// Initialize CUDA
	hipError_t cudaStatus = hipSetDevice(1);  // You can set the GPU device index as needed
	if (cudaStatus != hipSuccess)
		std::cerr << "hipSetDevice failed! Error: " << hipGetErrorString(cudaStatus) << std::endl;

	int device;
	hipGetDevice(&device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	unsigned short streams_size = deviceProp.asyncEngineCount;

	hipStream_t* streams = new hipStream_t[streams_size];
	for (unsigned short i = 0; i < streams_size; i++)
		hipStreamCreate(&streams[i]);

	// Creating the device variables
	double* d_camera_location, * d_vert,
		* d_V1, * d_V2, * d_V3,
		* d_t, * d_u, * d_v;

	bool* d_flag;

	// Allocate memory for device variables and copy data from host to device
	hipMalloc((void**)&d_camera_location, COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_vert, COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_V1, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V2, V_rows * COLUMNS_SIZE * sizeof(double));
	hipMalloc((void**)&d_V3, V_rows * COLUMNS_SIZE * sizeof(double));

	hipMalloc((void**)&d_flag, BLOCK_ROWS_SIZE * sizeof(bool));
	hipMalloc((void**)&d_t, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_u, BLOCK_ROWS_SIZE * sizeof(double));
	hipMalloc((void**)&d_v, BLOCK_ROWS_SIZE * sizeof(double));

	dim3 blockDim(1, BLOCK_ROWS_SIZE);
	dim3 gridDim(std::ceil(V_rows / BLOCK_ROWS_SIZE), 1);

	hipMemcpy(d_camera_location, camera_location, COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice);
	copy_V_arrays_to_device(V1, V2, V3, V_rows, d_V1, d_V2, d_V3);

	unsigned short stream = 0;
	for (unsigned short verts_row = 0; verts_row < verts_rows; verts_row++) {
		visible[verts_row] = true;
		stream = verts_row % streams_size;

		hipMemcpyAsync(d_vert, verts[verts_row], COLUMNS_SIZE * sizeof(double), hipMemcpyHostToDevice, streams[stream]);
		for (unsigned short V_row = 0, d_V_row = 0; V_row < V_rows - BLOCK_ROWS_SIZE + 1; V_row += BLOCK_ROWS_SIZE, d_V_row += V_BLOCK_SIZE) {
			//std::cout << "V_row = " << V_row << "\n";

			fastRayTriangleIntersection_parallel<<<gridDim, blockDim, 0, streams[stream]>>>(
				d_camera_location, d_vert,
				d_V1 + d_V_row, d_V2 + d_V_row, d_V3 + d_V_row, BLOCK_ROWS_SIZE,
				BORDER_EXCLUSIVE, LINE_TYPE_SEGMENT, PLANE_TYPE_TWOSIDED, false,
				d_flag, d_t, d_u, d_v);

			//std::cout << "After kernel, V_row = " << V_row << "\n";

			hipMemcpyAsync(flag + V_row, d_flag, BLOCK_ROWS_SIZE * sizeof(bool), hipMemcpyDeviceToHost, streams[stream]);
			//hipMemcpyAsync(t + V_row, d_t, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);
			//hipMemcpyAsync(u + V_row, d_u, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);
			//hipMemcpyAsync(v + V_row, d_v, BLOCK_ROWS_SIZE * sizeof(double), hipMemcpyDeviceToHost, streams[stream]);

			//std::cout << "After copying back all, V_row = " << V_row << "\n";

			for (unsigned short k = V_row; k < V_row + BLOCK_ROWS_SIZE; k++)
				if (flag[k]) {
					visible[verts_row] = false;
					break;
				}
		}

		if (verts_row % 100 == 0)
			std::cout << "Visible[" << verts_row << "] = " << visible[verts_row] << "\n";
	}

	hipFree(d_camera_location), hipFree(d_vert);
	hipFree(d_V1), hipFree(d_V2), hipFree(d_V3);
	hipFree(d_flag), hipFree(d_t), hipFree(d_u), hipFree(d_v);
}
